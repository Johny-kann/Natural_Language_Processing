#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "reductionCuda.cuh"

#include <stdio.h>
#include <string>
#include <iostream>

/*struct cudaString
{
	char *str;
	int length;
};

struct myString
{
	char *str;
	short index[10];
	short totIndexes;
};
*/

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t testWithCuda(unsigned int size);

__device__ void wordHandlerDevice(char *c, int *indexes, int lineNum)
{
	char prev = '0', curr;
	indexes[0] = 150 * lineNum;

	int count = 0;

	for (int i = 0; c[i] != '\0'; i++)
	{
		curr = c[i];
		if (i != 0)
			prev = c[i - 1];

		if (!((curr >= 'A' && curr <= 'Z') || (curr >= 'a' && curr <= 'z') || (curr >= '0' && curr <= '9')))
		{
			c[i] = '\0';
			curr = '\0';
		}

		if (prev == '\0' && curr != '\0')
		{
			count++;
			indexes[count] = 150 * lineNum + i;
		}
	}

	count++;
	indexes[count] = -1;
}

__device__ void charLineHandlerDevice(char *c, int *indexes, unsigned int lineNum)
{
	char prev, prev2, curr;
//	indexes[0] = 300 * lineNum;

	int count = 0;

//	printf("kernal %d", blockIdx.x*blockDim.x+threadIdx.x);

	int i;
	for (i = 2; c[i] != '\0'; i++)
	{
		//	c[i] = c[i] + 1;
		//	printf("%c", c[i]);
		curr = c[i];
		prev = c[i - 1];
		prev2 = c[i - 2];

		if (curr == '\"' && prev == ',' && prev2 == '\"')
		{
			c[i] = '\0';
			c[i - 1] = '\0';
			c[i - 2] = '\0';
		}
		else if (prev == '\0')
		{
			count++;
			indexes[count] = 300 * lineNum + i;
		}

		if (curr >= 'A' && curr <= 'Z')
			c[i]=c[i]+32;

	}

		c[0] = '\0';
		c[i-1] = '\0';
		indexes[0] = 300 * lineNum + 1;
		count++;
		indexes[count] = -1;
}

__global__ void tweetToWordKernal(char *message, int *index, unsigned int size)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;

	if (i<size)
		wordHandlerDevice(&message[150 * i], &index[70 * i], i);
	
}

__global__ void separateKernal(char *charIn, int *indexes, unsigned int size)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;

//	__syncthreads();
	if (i < size)
	{
		charLineHandlerDevice(charIn + i * 300, indexes + i * 20, i);

	//	printf("%d ", i);
	}
//	__syncthreads();


}




__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void testKernel(cudaString str)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
//	str.at(0);
//	std::cout << str;
	printf("%s\t",str.str);
}


int main4()
{
	

/*	hipError_t cudaStatus = testWithCuda(10);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}*/
	


	getchar();

	return 0;
}

int mainK()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t separateStub(char *chars, int *indexes, unsigned int size);

hipError_t tweetToWordStub(char *chars, int *indexes, unsigned int size);

int tweetToWordCuda(char *message, int *index, unsigned int size)
{
	// Add vectors in parallel.
	hipError_t cudaStatus = tweetToWordStub(message, index, size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

int separator(char *chars, int *indexes, unsigned int size)
{
	// Add vectors in parallel.
	hipError_t cudaStatus = separateStub(chars, indexes, size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

hipError_t tweetToWordStub(char *chars, int *indexes, unsigned int size)
{
	char *dev_in;
	int *dev_out;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output).

//	std::cout << 150 * size*sizeof(char) << '\t';
	cudaStatus = hipMalloc((void**)&dev_in, 150 * size * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .

//	std::cout << 70 * size*sizeof(int) << '\t';
	cudaStatus = hipMalloc((void**)&dev_out, 70 * size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_in, chars, 150 * size * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda String hipMemcpy failed!");
		goto Error;
	}

	dim3 grid;
	dim3 block;

	block.x = 1024;
	block.y = 1;
	block.z = 1;
	grid.x = ceil((double)size / 1024);
	grid.y = 1;
	grid.z = 1;

	std::cout << grid.x << block.x;

	tweetToWordKernal << < grid, block >> >(dev_in, dev_out, size);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "separateKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(indexes, dev_out, 70 * size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Indexes Cuda tweet to word hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(chars, dev_in, 150 * size * sizeof(char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "chars Cuda tweet to word hipMemcpy failed!");
		goto Error;
	}

	
Error:
	hipFree(dev_in);
	hipFree(dev_out);

	return cudaStatus;
}

hipError_t separateStub(char *chars, int *indexes, unsigned int size)
{
	char *dev_in;
	int *dev_out;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output).

	std::cout << 300 * size*sizeof(char)<<'\t';
	cudaStatus = hipMalloc((void**)&dev_in, 300*size * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .

	std::cout << 20 * size*sizeof(int) << '\t';
	cudaStatus = hipMalloc((void**)&dev_out, 20*size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_in, chars, 300*size * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda String hipMemcpy failed!");
		goto Error;
	}

	dim3 grid;
	dim3 block;

	block.x = 512;
	block.y = 1;
	block.z = 1;
	grid.x = ceil((double)size / 512);
	grid.y = 1;
	grid.z = 1;

	std::cout << grid.x << block.x;

		separateKernal << < grid, block >> >(dev_in,dev_out,size);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "separateKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(indexes, dev_out, 20 * size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "tweets Cuda hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(chars, dev_in, 300 * size * sizeof(char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "tweets Cuda hipMemcpy failed!");
		goto Error;
	}

//	for (int i = 0; i < 300*size; i++)
//		std::cout << chars[i];

//	for (int i = 0; i < 20 * size; i++)
//		std::cout << indexes[i]<<' ';
Error:
	hipFree(dev_in);
	hipFree(dev_out);

	return cudaStatus;
}

hipError_t testWithCuda(unsigned int size)
{
/*	std::string str = "Hello";

	cudaString string;

	string.str = new char[str.size() + 1];

	memcpy(string.str, str.c_str(), str.size());
	*/

	char *strr = "Hello";
	cudaString str;
	str.str = strr;

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	

	// Launch a kernel on the GPU with one thread for each element.
	testKernel << <1, size >> >(str);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.

Error:

	return cudaStatus;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
