#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "reductionCuda.cuh"

#include <stdio.h>
#include <string>
#include <iostream>

/*struct cudaString
{
	char *str;
	int length;
};

struct myString
{
	char *str;
	short index[10];
	short totIndexes;
};
*/

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t testWithCuda(unsigned int size);

__global__ void separateKernal(cudaString *cuda, myString *temp, unsigned int size)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;

	if (i < size)
	{
	char *prev, *prev2, *curr;
	temp[i].str = cuda[i].str;
	char *c = cuda[i].str;
	temp[i].index[0] = 0;
	temp[i].totIndexes = 1;

	for (int i = 2; i < cuda[i].length; i++)
	{
		curr = &c[i];
		prev = &c[i - 1];
		prev2 = &c[i - 2];

		if (*curr == '\"' && *prev == ',' && *prev2 == '\"')
		{

			c[i] = '\0';

			c[i - 1] = '\0';

			c[i - 2] = '\0';

		}
		else if (*prev == '\0')
		{
			temp->index[temp->totIndexes] = i;
			temp->totIndexes++;
		}
	}

	c[0] = '\0';
	temp[i].index[0] = 1;
	c[cuda[i].length - 1] = '\0';

	}

}



__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void testKernel(cudaString str)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
//	str.at(0);
//	std::cout << str;
	printf("%s\t",str.str);
}


int main4()
{
	

/*	hipError_t cudaStatus = testWithCuda(10);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}*/
	


	getchar();

	return 0;
}

int main1()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

int separator(cudaString *linesCuda, myString *tweetsCuda, unsigned int size)
{
	// Add vectors in parallel.
	hipError_t cudaStatus = separateStub(linesCuda, tweetsCuda, size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

hipError_t separateStub(cudaString *linesCuda, myString *tweetsCuda, unsigned int size)
{
	cudaString *dev_in;
	myString *dev_out;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_in, size * sizeof(cudaString));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_out, size * sizeof(myString));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_in, linesCuda, size * sizeof(cudaString), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda String hipMemcpy failed!");
		goto Error;
	}

	dim3 grid;
	dim3 block;

	block.x = 512;
	block.y = 0;
	block.z = 0;
	grid.x = ceil(size / 512);
	grid.y = 0;
	grid.z = 0;

		separateKernal << < grid, block >> >(dev_in,dev_out,size);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "separateKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(tweetsCuda, dev_out, size * sizeof(myString), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "tweets Cuda hipMemcpy failed!");
		goto Error;
	}

	

/*	for (int i = 0; i < size; i++)
	{
		separate(&linesCuda[i], &tweetsCuda[i]);
		//	linesCuda[i].str, linesCuda[i].length, &tweetsCuda[i]);
	}
	*/

Error:
	hipFree(dev_in);
	hipFree(dev_out);

	return cudaStatus;
}

hipError_t testWithCuda(unsigned int size)
{
/*	std::string str = "Hello";

	cudaString string;

	string.str = new char[str.size() + 1];

	memcpy(string.str, str.c_str(), str.size());
	*/

	char *strr = "Hello";
	cudaString str;
	str.str = strr;

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	

	// Launch a kernel on the GPU with one thread for each element.
	testKernel << <1, size >> >(str);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.

Error:

	return cudaStatus;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
