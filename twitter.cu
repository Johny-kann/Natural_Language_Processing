#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "reductionCuda.cuh"

#include <stdio.h>
#include <string>
#include <iostream>

/*struct cudaString
{
	char *str;
	int length;
};

struct myString
{
	char *str;
	short index[10];
	short totIndexes;
};
*/

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t testWithCuda(unsigned int size);

__device__ void wordHandlerDevice(char *c, int *indexes, int lineNum)
{
	char prev = '0', curr;
	indexes[0] = 150 * lineNum;

	int count = 0;

	for (int i = 0; c[i] != '\0'; i++)
	{
		curr = c[i];
		if (i != 0)
			prev = c[i - 1];

		if (!((curr >= 'A' && curr <= 'Z') || (curr >= 'a' && curr <= 'z') || (curr >= '0' && curr <= '9')))
		{
			c[i] = '\0';
			curr = '\0';
		}

		if (prev == '\0' && curr != '\0')
		{
			count++;
			indexes[count] = 150 * lineNum + i;
		}
	}

	count++;
	indexes[count] = -1;
}


__device__ __host__ int stringCompare(char *source, char *dest)
{
	int i;
	for (i = 0; source[i] != '\0'; i++)
	{
		if (dest[i] > source[i])
			return 1;
		else if (dest[i] < source[i])
			return -1;
	}

	if (dest[i] != '\0')
		return 1;

	return 0;
}

__device__ void compareWords(char *tweet, int *posNeg, char *posWords, unsigned int posWordSize, char *negWords, unsigned int negWordSize)
{
	int indexes[50];
	char prev = '0', curr;
	indexes[0] = 0;

	int count = 0;

	for (int i = 0; tweet[i] != '\0'; i++)
	{
		curr = tweet[i];
		if (i != 0)
			prev = tweet[i - 1];

		if (!((curr >= 'A' && curr <= 'Z') || (curr >= 'a' && curr <= 'z') || (curr >= '0' && curr <= '9')))
		{
			tweet[i] = '\0';
			curr = '\0';
		}

		if (prev == '\0' && curr != '\0')
		{
			count++;
			indexes[count] = i;
		}
	}

	count++;
	indexes[count] = -1;

	int prob = 0;

	for (int i = 0; indexes[i] != -1; i++)
	{
		for (int j = 0; j < posWordSize; j++)
		{
			if (stringCompare(&tweet[indexes[i]], &posWords[25*j]) == 0)
			{
			//	printf("\nPositive Prob for word %s in thread %d", &tweet[indexes[i]], threadIdx.x);
				prob++; break;
			}
			
		}

		for (int j = 0; j < negWordSize; j++)
		{
			if (stringCompare(&tweet[indexes[i]], &negWords[25*j]) == 0)
			{
				prob--; break;
			}
		}
	}

	if (prob > 0)
	{
		*posNeg = 1;
	}
	else if (prob < 0)
	{
		*posNeg = -1;
	}
	else
	{
		*posNeg = 0;
	}
}

__device__ void charLineHandlerDevice(char *c, int *indexes, unsigned int lineNum)
{
	char prev, prev2, curr;
//	indexes[0] = 300 * lineNum;

	int count = 0;

//	printf("kernal %d", blockIdx.x*blockDim.x+threadIdx.x);

	int i;
	for (i = 2; c[i] != '\0'; i++)
	{
		//	c[i] = c[i] + 1;
		//	printf("%c", c[i]);
		curr = c[i];
		prev = c[i - 1];
		prev2 = c[i - 2];

		if (curr == '\"' && prev == ',' && prev2 == '\"')
		{
			c[i] = '\0';
			c[i - 1] = '\0';
			c[i - 2] = '\0';
		}
		else if (prev == '\0')
		{
			count++;
			indexes[count] = 300 * lineNum + i;
		}

		if (curr >= 'A' && curr <= 'Z')
			c[i]=c[i]+32;

	}

		c[0] = '\0';
		c[i-1] = '\0';
		indexes[0] = 300 * lineNum + 1;
		count++;
		indexes[count] = -1;
}

__global__ void tweetToWordKernal(char *message, int *index, unsigned int size)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;

	if (i<size)
		wordHandlerDevice(&message[150 * i], &index[70 * i], i);
	
}

__global__ void tweetToProbKernal(char *tweets, int *prob, unsigned int size,char *posWords, unsigned int posWordSize, char *negWords, unsigned int negWordSize)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;

	if (i < size)
	{	
		compareWords(&tweets[150 * i], &prob[i], posWords, posWordSize, negWords, negWordSize);

	}

}

__global__ void separateKernal(char *charIn, int *indexes, unsigned int size)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;

//	__syncthreads();
	if (i < size)
	{
		charLineHandlerDevice(charIn + i * 300, indexes + i * 20, i);

	//	printf("%d ", i);
	}
//	__syncthreads();


}



hipError_t separateStub(char *chars, int *indexes, unsigned int size);

hipError_t tweetToWordStub(char *chars, int *indexes, unsigned int size);

hipError_t tweetToProbStub(char *tweets, int *prob, int size, char *posWords, int posWordSize, char *negWords, int negWordSize);

int tweetToProb(char *tweets, int *prob, int size, char *posWords, int posWordSize, char *negWords, int negWordSize)
{
	hipError_t cudaStatus = tweetToProbStub(tweets, prob, size, posWords, posWordSize, negWords, negWordSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

int tweetToWordCuda(char *message, int *index, unsigned int size)
{
	// Add vectors in parallel.
	hipError_t cudaStatus = tweetToWordStub(message, index, size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

int separator(char *chars, int *indexes, unsigned int size)
{
	// Add vectors in parallel.
	hipError_t cudaStatus = separateStub(chars, indexes, size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

hipError_t tweetToProbStub(char *tweets, int *prob, int size, char *posWords, int posWordSize, char *negWords, int negWordSize)
{
	char *dev_tweet_in;
	int *dev_prob_out;

	char *dev_posWords_in;
	char *dev_negWords_in;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_tweet_in, 150 * size * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_prob_out, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_posWords_in, 25 * posWordSize * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_negWords_in, 25 * negWordSize * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_tweet_in, tweets, 150 * size * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda String hipMemcpy failed!");
		goto Error;
	}

//	for (int i = 0; i < 3; i++)
//		printf("%s ", &posWords[i * 25]);

	cudaStatus = hipMemcpy(dev_posWords_in, posWords, 25 * posWordSize * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda String hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_negWords_in, negWords, 25 * negWordSize * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda String hipMemcpy failed!");
		goto Error;
	}

	/*---------------------------Kernal----------------------*/
	dim3 grid;
	dim3 block;

	block.x = 1024;
	block.y = 1;
	block.z = 1;

	grid.x = ceil((double)size / block.x);
	grid.y = 1;
	grid.z = 1;

	
	tweetToProbKernal << < grid, block >> >(dev_tweet_in , dev_prob_out , size, dev_posWords_in , posWordSize , dev_negWords_in , negWordSize);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "tweetToProbKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching tweetToProbKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(prob, dev_prob_out ,size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Prob hipMemcpy failed!");
		goto Error;
	}


Error:
	hipFree(dev_negWords_in);
	hipFree(dev_posWords_in);
	hipFree(dev_prob_out);
	hipFree(dev_tweet_in);
//	hipFree(dev_);

	return cudaStatus;

}

hipError_t tweetToWordStub(char *chars, int *indexes, unsigned int size)
{
	char *dev_in;
	int *dev_out;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output).

//	std::cout << 150 * size*sizeof(char) << '\t';
	cudaStatus = hipMalloc((void**)&dev_in, 150 * size * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .

//	std::cout << 70 * size*sizeof(int) << '\t';
	cudaStatus = hipMalloc((void**)&dev_out, 70 * size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_in, chars, 150 * size * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda String hipMemcpy failed!");
		goto Error;
	}

	dim3 grid;
	dim3 block;

	block.x = 1024;
	block.y = 1;
	block.z = 1;
	grid.x = ceil((double)size / 1024);
	grid.y = 1;
	grid.z = 1;

	std::cout << grid.x << block.x;

	tweetToWordKernal << < grid, block >> >(dev_in, dev_out, size);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "separateKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(indexes, dev_out, 70 * size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Indexes Cuda tweet to word hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(chars, dev_in, 150 * size * sizeof(char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "chars Cuda tweet to word hipMemcpy failed!");
		goto Error;
	}

	
Error:
	hipFree(dev_in);
	hipFree(dev_out);

	return cudaStatus;
}

hipError_t separateStub(char *chars, int *indexes, unsigned int size)
{
	char *dev_in;
	int *dev_out;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output).

	std::cout << 300 * size*sizeof(char)<<'\t';
	cudaStatus = hipMalloc((void**)&dev_in, 300*size * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .

	std::cout << 20 * size*sizeof(int) << '\t';
	cudaStatus = hipMalloc((void**)&dev_out, 20*size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_in, chars, 300*size * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda String hipMemcpy failed!");
		goto Error;
	}

	dim3 grid;
	dim3 block;

	block.x = 512;
	block.y = 1;
	block.z = 1;
	grid.x = ceil((double)size / 512);
	grid.y = 1;
	grid.z = 1;

	std::cout << grid.x << block.x;

		separateKernal << < grid, block >> >(dev_in,dev_out,size);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "separateKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(indexes, dev_out, 20 * size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "tweets Cuda hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(chars, dev_in, 300 * size * sizeof(char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "tweets Cuda hipMemcpy failed!");
		goto Error;
	}

//	for (int i = 0; i < 300*size; i++)
//		std::cout << chars[i];

//	for (int i = 0; i < 20 * size; i++)
//		std::cout << indexes[i]<<' ';
Error:
	hipFree(dev_in);
	hipFree(dev_out);

	return cudaStatus;
}


